#include "hip/hip_runtime.h"
#include "faster_faster.cuh"

#define CHECK_CUDA_ERROR(val) check((val), __FILE__, __LINE__)
template <typename T>
void check(T err, const char* file, const int line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

__global__ void increment(int8_t* ptr);

int open_with_cufile(){
    int f_desc = -1;
    int ret = -1;

    CUfileError_t cf_stat;
    CUfileDescr_t cf_desc;
    CUfileHandle_t cf_handle;

    const char* file_path = "/home/ben/Desktop/direct-storage/demo.txt";

    f_desc = open(file_path, O_CREAT | O_RDWR | O_DIRECT); /*API requires O_DIRECT MODE*/
    if (f_desc < 0){
        std::cerr << "The file has not been opened..ERROR" << std::endl;
        return -1;
    }else{
        std::cout << "File has been opened properly..PASSED" << std::endl;
    }

    struct stat file_stats;
    ret = fstat(f_desc,&file_stats);
    if(ret == -1){
        std::cout << "File stats has not been occured..ERROR" << std::endl;
    }else{
        std::cout << "The file is : "<< file_stats.st_size <<" bytes..PASSED" << std::endl;
    }

    size_t size_of_file = file_stats.st_size;

    memset((void *)&cf_desc,0,sizeof(CUfileDescr_t));

    cf_desc.handle.fd = f_desc;
    cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD; /*That means this is linux based file*/
    cf_stat = cuFileHandleRegister(&cf_handle,&cf_desc);
    if(cf_stat.err == CU_FILE_SUCCESS){
        std::cout << "File has been handled successfully..PASSED" << std::endl;
    }else{
        std::cerr << "The file has not been opened..ERROR" << std::endl;
        return -1;
    }

    int8_t* dev_ptr = nullptr;
    CHECK_CUDA_ERROR(hipMalloc((int8_t**)&dev_ptr,size_of_file));
    CHECK_CUDA_ERROR(hipMemset((int8_t*)dev_ptr,0,size_of_file));
    CHECK_CUDA_ERROR(hipStreamSynchronize(0)); /*wait until operations are done*/

    ret = cuFileRead(cf_handle,dev_ptr,size_of_file,0,0); /*returns size of bytes successfully written*/
    if(ret < 0){
        std::cerr << "Something went wrong while reading..ERROR" << ret << std::endl;
    }else{
        std::cout << "Read bytes :" << ret << "..PASSED" <<std::endl;
    }
    increment<<<1,9>>>((int8_t *)dev_ptr);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    ret = cuFileWrite(cf_handle,dev_ptr,size_of_file,0,0); /*returns size of bytes successfully written*/
    if(ret < 0){
        std::cerr << "Something went wrong while writing..ERROR" << ret << std::endl;
    }else{
        std::cout << "Written bytes :" << ret << "..PASSED" <<std::endl;
    }

    CHECK_CUDA_ERROR(hipFree(dev_ptr));
    cuFileHandleDeregister(cf_handle);
    close(f_desc);
    return 0;
}

__global__ void increment(int8_t* ptr){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    ptr[threadId]+=1;
}